#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#define CHECK_CUDA_ERROR(call) \
	do { \
		hipError_t error = call; \
		if (error != hipSuccess) { \
			std::cerr << "CUDA Error: " << hipGetErrorString(error) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
			exit(EXIT_FAILURE); \
		} \
	} while (0)

int main() {
	const size_t size = 1024 * 1024 * 1024; // 100 MB
	float *h_data = (float*)malloc(size);
	float *d_data;

	CHECK_CUDA_ERROR(hipMalloc((void**)&d_data, size));

	hipEvent_t start, stop;
	CHECK_CUDA_ERROR(hipEventCreate(&start));
	CHECK_CUDA_ERROR(hipEventCreate(&stop));

    std::cout << "begin transfer" << '\n';
	CHECK_CUDA_ERROR(hipEventRecord(start, 0));
    int times = 100;
    for (int i = 0; i < times; i++) {
        CHECK_CUDA_ERROR(hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice));
    }
	CHECK_CUDA_ERROR(hipEventRecord(stop, 0));

	CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    std::cout << "end transfer" << '\n';

	float milliseconds = 0;
	CHECK_CUDA_ERROR(hipEventElapsedTime(&milliseconds, start, stop));

	float bandwidth = times * (size / (1 << 20)) / (milliseconds / 1000.0f);
	std::cout << "Host to Device Bandwidth: " << bandwidth << " MB/s" << std::endl;
    std::cout << "Transfer time: " << milliseconds / 1000.0f << " s" << std::endl;
    std::cout << "Transfer size: " << times * (size / (1 << 20)) << " MB" << std::endl;

	CHECK_CUDA_ERROR(hipEventDestroy(start));
	CHECK_CUDA_ERROR(hipEventDestroy(stop));
	CHECK_CUDA_ERROR(hipFree(d_data));
	free(h_data);
	return 0;
}
